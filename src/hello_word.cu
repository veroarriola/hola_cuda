#include "hip/hip_runtime.h"
// Referencias:
// https://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf

#include <stdio.h>
#define N 512

__global__ void helloCUDA()
{
    printf("Hello, from CUDA!\n");
}

__global__ void add(int *a, int *b, int *c)
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main()
{
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();

    /// 
    /// Memoria
    ///

    int *a, *b, *c;           // cpu anfitrión (host)
    int *d_a, *d_b, *d_c;     // tarjeta dispositivo(device)
    int size = N * sizeof(int);

    // Solicitar espacio en la tarjeta
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Asignar espacio en el anfitrión
    // Llenar los vectores con valores aleatorios
    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b ,N);
    // Espacio para el resultado
    c = (int *)malloc(size);


    ///
    /// Transferencia
    ///

    // Enviar datos a la tarjeta
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Ejecutar el kernel en la tarjeta con N bloques
    add<<<N,1>>>(d_a, d_b, d_c);

    // Copiar el resultado de regreso al anfitrión
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);


    ///
    /// Limpieza
    ///
    free(a); free(b); free(c);                    // cpu
    hipFree(d_a); hipFree(d_b); hipFree(d_c);  // tarjeta

    return 0;
}

